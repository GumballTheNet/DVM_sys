#include "hip/hip_runtime.h"

#include <dvmhlib_cuda.h>
#define dcmplx2 Complex<double>
#define cmplx2 Complex<float>
typedef int __indexTypeInt; 
typedef long long __indexTypeLLong;





//--------------------- Kernel for loop on line 49 ---------------------

      __global__ void   loop_test2_49_cuda_kernel_int(float b[], __indexTypeInt b0002, float a[], __indexTypeInt a0002, __indexTypeInt begin_1, __indexTypeInt end_1, __indexTypeInt begin_2, __indexTypeInt end_2, __indexTypeInt blocks_1, __indexTypeInt add_blocks, int l)
      {

// Local needs
         int i;
         int j;
         __indexTypeInt rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               a[i + a0002 * j] = 0.f;
               if (i == 1 | j == 1 | i == l | j == l) 
               {
                  b[i + b0002 * j] = 0.f;
               }
               else 
               {
                  b[i + b0002 * j] = 1.f + i + j;
               }
            }
         }
      }


//--------------------- Kernel for loop on line 49 ---------------------

      __global__ void   loop_test2_49_cuda_kernel_llong(float b[], __indexTypeLLong b0002, float a[], __indexTypeLLong a0002, __indexTypeLLong begin_1, __indexTypeLLong end_1, __indexTypeLLong begin_2, __indexTypeLLong end_2, __indexTypeLLong blocks_1, __indexTypeLLong add_blocks, int l)
      {

// Local needs
         int i;
         int j;
         __indexTypeLLong rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               a[i + a0002 * j] = 0.f;
               if (i == 1 | j == 1 | i == l | j == l) 
               {
                  b[i + b0002 * j] = 0.f;
               }
               else 
               {
                  b[i + b0002 * j] = 1.f + i + j;
               }
            }
         }
      }


//--------------------- Kernel for loop on line 65 ---------------------

      __global__ void   loop_test2_65_cuda_kernel_int(float a[], __indexTypeInt a0002, float b[], __indexTypeInt b0002, float eps, float eps_grid[], __indexTypeInt begin_1, __indexTypeInt end_1, __indexTypeInt begin_2, __indexTypeInt end_2, __indexTypeInt blocks_1, __indexTypeInt add_blocks)
      {

// Local needs
         int i;
         int j;
         __indexTypeInt rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               eps = max(eps, abs(b[i + b0002 * j] - a[i + a0002 * j]));
               a[i + a0002 * j] = b[i + b0002 * j];
            }
         }

// Reduction
         i = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y);
         eps = __dvmh_blockReduceMax(eps);
         if (i % warpSize == 0) 
         {
            eps_grid[(add_blocks + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z / warpSize) + i / warpSize] = eps;
         }
      }


//--------------------- Kernel for loop on line 65 ---------------------

      __global__ void   loop_test2_65_cuda_kernel_llong(float a[], __indexTypeLLong a0002, float b[], __indexTypeLLong b0002, float eps, float eps_grid[], __indexTypeLLong begin_1, __indexTypeLLong end_1, __indexTypeLLong begin_2, __indexTypeLLong end_2, __indexTypeLLong blocks_1, __indexTypeLLong add_blocks)
      {

// Local needs
         int i;
         int j;
         __indexTypeLLong rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               eps = max(eps, abs(b[i + b0002 * j] - a[i + a0002 * j]));
               a[i + a0002 * j] = b[i + b0002 * j];
            }
         }

// Reduction
         i = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y);
         eps = __dvmh_blockReduceMax(eps);
         if (i % warpSize == 0) 
         {
            eps_grid[(add_blocks + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z / warpSize) + i / warpSize] = eps;
         }
      }


//--------------------- Kernel for loop on line 73 ---------------------

      __global__ void   loop_test2_73_cuda_kernel_int(float b[], __indexTypeInt b0002, float a[], __indexTypeInt a0002, __indexTypeInt begin_1, __indexTypeInt end_1, __indexTypeInt begin_2, __indexTypeInt end_2, __indexTypeInt blocks_1, __indexTypeInt add_blocks)
      {

// Local needs
         int i;
         int j;
         __indexTypeInt rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               b[i + b0002 * j] = (a[i - 1 + a0002 * j] + a[i + a0002 * (j - 1)] + a[i + 1 + a0002 * j] + a[i + a0002 * (j + 1)]) / 4;
            }
         }
      }


//--------------------- Kernel for loop on line 73 ---------------------

      __global__ void   loop_test2_73_cuda_kernel_llong(float b[], __indexTypeLLong b0002, float a[], __indexTypeLLong a0002, __indexTypeLLong begin_1, __indexTypeLLong end_1, __indexTypeLLong begin_2, __indexTypeLLong end_2, __indexTypeLLong blocks_1, __indexTypeLLong add_blocks)
      {

// Local needs
         int i;
         int j;
         __indexTypeLLong rest_blocks, cur_blocks;

// Calculate each thread's loop variables' values
         rest_blocks = add_blocks + blockIdx.x;
         cur_blocks = rest_blocks / blocks_1;
         j = begin_1 + (cur_blocks * blockDim.y + threadIdx.y);
         if (j <= end_1) 
         {
            rest_blocks = rest_blocks - cur_blocks * blocks_1;
            cur_blocks = rest_blocks;
            i = begin_2 + (cur_blocks * blockDim.x + threadIdx.x);
            if (i <= end_2) 
            {

// Loop body
               b[i + b0002 * j] = (a[i - 1 + a0002 * j] + a[i + a0002 * (j - 1)] + a[i + 1 + a0002 * j] + a[i + a0002 * (j + 1)]) / 4;
            }
         }
      }



#ifdef _MS_F_
#define loop_test2_49_cuda_ loop_test2_49_cuda
#define loop_test2_65_cuda_ loop_test2_65_cuda
#define loop_test2_73_cuda_ loop_test2_73_cuda
#endif

extern "C" {
      extern  DvmType loop_test2_73_cuda_kernel_llong_regs, loop_test2_73_cuda_kernel_int_regs, loop_test2_65_cuda_kernel_llong_regs, loop_test2_65_cuda_kernel_int_regs, loop_test2_49_cuda_kernel_llong_regs, loop_test2_49_cuda_kernel_int_regs;


//    CUDA handler for loop on line 49 

      void   loop_test2_49_cuda_(DvmType *loop_ref, DvmType b[], DvmType a[], int *l)
      {
         void   *b_base, *a_base;
         DvmType d_b[5], d_a[5];
         DvmType idxTypeInKernel;
         dim3 blocks, threads;
         hipStream_t stream;
         DvmType idxL[2], idxH[2];
         DvmType blocksS[2], restBlocks, maxBlocks, addBlocks, overallBlocks;
         DvmType device_num;

// Get device number
         device_num = loop_get_device_num_(loop_ref);

// Get 'natural' bases
         b_base = dvmh_get_natural_base(&device_num, b);
         a_base = dvmh_get_natural_base(&device_num, a);

// Fill 'device' headers
         dvmh_fill_header_(&device_num, b_base, b, d_b);
         dvmh_fill_header_(&device_num, a_base, a, d_a);

// Guess index type in CUDA kernel
         idxTypeInKernel = loop_guess_index_type_(loop_ref);
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(int)) 
         {
            idxTypeInKernel = rt_INT;
         }
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(long long)) 
         {
            idxTypeInKernel = rt_LLONG;
         }

// Get CUDA configuration parameters
         threads = dim3(0, 0, 0);
         if (idxTypeInKernel == rt_INT) 
         {
            loop_cuda_get_config(loop_ref, 0, loop_test2_49_cuda_kernel_int_regs, &threads, &stream, 0);
         }
         else 
         {
            loop_cuda_get_config(loop_ref, 0, loop_test2_49_cuda_kernel_llong_regs, &threads, &stream, 0);
         }
         loop_fill_bounds_(loop_ref, idxL, idxH, 0);
         blocksS[1] = (idxH[1] - idxL[1] + threads.x) / threads.x;
         blocksS[0] = blocksS[1] * ((idxH[0] - idxL[0] + threads.y) / threads.y);
         overallBlocks = blocksS[0];
         restBlocks = overallBlocks;
         addBlocks = 0;
         blocks = dim3(1, 1, 1);
         maxBlocks = loop_cuda_get_device_prop(loop_ref, CUDA_MAX_GRID_X);

// GPU execution
         while (restBlocks > 0)
         {
            if (restBlocks <= maxBlocks) 
            {
               blocks = restBlocks;
            }
            else 
            {
               blocks = maxBlocks;
            }
            if (idxTypeInKernel == rt_INT) 
            {
               loop_test2_49_cuda_kernel_int<<<blocks, threads, 0, stream>>>((float *)b_base, d_b[1], (float *)a_base, d_a[1], idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks, *l);
            }
            else 
            {
               loop_test2_49_cuda_kernel_llong<<<blocks, threads, 0, stream>>>((float *)b_base, d_b[1], (float *)a_base, d_a[1], idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks, *l);
            }
            addBlocks += blocks.x;
            restBlocks -= blocks.x;
         }
      }


//    CUDA handler for loop on line 65 

      void   loop_test2_65_cuda_(DvmType *loop_ref, DvmType a[], DvmType b[])
      {
         void   *a_base, *b_base;
         DvmType d_a[5], d_b[5];
         DvmType idxTypeInKernel;
         dim3 blocks, threads;
         hipStream_t stream;
         DvmType idxL[2], idxH[2];
         DvmType blocksS[2], restBlocks, maxBlocks, addBlocks, overallBlocks;
         void   *eps_grid;
         float eps;
         DvmType red_num, num_of_red_blocks, fill_flag;
         DvmType shared_mem;
         DvmType device_num;

// Get device number
         device_num = loop_get_device_num_(loop_ref);

// Register reduction for CUDA-execution
         red_num = 1;
         loop_cuda_register_red(loop_ref, red_num, &eps_grid, 0);
         loop_red_init_(loop_ref, &red_num, &eps, 0);

// Get 'natural' bases
         a_base = dvmh_get_natural_base(&device_num, a);
         b_base = dvmh_get_natural_base(&device_num, b);

// Fill 'device' headers
         dvmh_fill_header_(&device_num, a_base, a, d_a);
         dvmh_fill_header_(&device_num, b_base, b, d_b);

// Guess index type in CUDA kernel
         idxTypeInKernel = loop_guess_index_type_(loop_ref);
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(int)) 
         {
            idxTypeInKernel = rt_INT;
         }
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(long long)) 
         {
            idxTypeInKernel = rt_LLONG;
         }

// Get CUDA configuration parameters
         threads = dim3(0, 0, 0);
#ifdef CUDA_FERMI_ARCH
         shared_mem = 4;
#else
         shared_mem = 0;
#endif
         if (idxTypeInKernel == rt_INT) 
         {
            loop_cuda_get_config(loop_ref, shared_mem, loop_test2_65_cuda_kernel_int_regs, &threads, &stream, &shared_mem);
         }
         else 
         {
            loop_cuda_get_config(loop_ref, shared_mem, loop_test2_65_cuda_kernel_llong_regs, &threads, &stream, &shared_mem);
         }
         loop_fill_bounds_(loop_ref, idxL, idxH, 0);
         blocksS[1] = (idxH[1] - idxL[1] + threads.x) / threads.x;
         blocksS[0] = blocksS[1] * ((idxH[0] - idxL[0] + threads.y) / threads.y);
         overallBlocks = blocksS[0];
         restBlocks = overallBlocks;
         addBlocks = 0;
         blocks = dim3(1, 1, 1);

// Prepare reduction
         num_of_red_blocks = overallBlocks * (threads.x * threads.y * threads.z / 32);
         fill_flag = 0;
         red_num = 1;
         loop_cuda_red_prepare(loop_ref, red_num, num_of_red_blocks, fill_flag);
         maxBlocks = loop_cuda_get_device_prop(loop_ref, CUDA_MAX_GRID_X);

// GPU execution
         while (restBlocks > 0)
         {
            if (restBlocks <= maxBlocks) 
            {
               blocks = restBlocks;
            }
            else 
            {
               blocks = maxBlocks;
            }
            if (idxTypeInKernel == rt_INT) 
            {
               loop_test2_65_cuda_kernel_int<<<blocks, threads, shared_mem, stream>>>((float *)a_base, d_a[1], (float *)b_base, d_b[1], eps, (float *)eps_grid, idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks);
            }
            else 
            {
               loop_test2_65_cuda_kernel_llong<<<blocks, threads, shared_mem, stream>>>((float *)a_base, d_a[1], (float *)b_base, d_b[1], eps, (float *)eps_grid, idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks);
            }
            addBlocks += blocks.x;
            restBlocks -= blocks.x;
         }

// Finish reduction
         red_num = 1;
         loop_red_finish(loop_ref, red_num);
      }


//    CUDA handler for loop on line 73 

      void   loop_test2_73_cuda_(DvmType *loop_ref, DvmType b[], DvmType a[])
      {
         void   *b_base, *a_base;
         DvmType d_b[5], d_a[5];
         DvmType idxTypeInKernel;
         dim3 blocks, threads;
         hipStream_t stream;
         DvmType idxL[2], idxH[2];
         DvmType blocksS[2], restBlocks, maxBlocks, addBlocks, overallBlocks;
         DvmType device_num;

// Get device number
         device_num = loop_get_device_num_(loop_ref);

// Get 'natural' bases
         b_base = dvmh_get_natural_base(&device_num, b);
         a_base = dvmh_get_natural_base(&device_num, a);

// Fill 'device' headers
         dvmh_fill_header_(&device_num, b_base, b, d_b);
         dvmh_fill_header_(&device_num, a_base, a, d_a);

// Guess index type in CUDA kernel
         idxTypeInKernel = loop_guess_index_type_(loop_ref);
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(int)) 
         {
            idxTypeInKernel = rt_INT;
         }
         if (idxTypeInKernel == rt_LONG && sizeof(long) == sizeof(long long)) 
         {
            idxTypeInKernel = rt_LLONG;
         }

// Get CUDA configuration parameters
         threads = dim3(0, 0, 0);
         if (idxTypeInKernel == rt_INT) 
         {
            loop_cuda_get_config(loop_ref, 0, loop_test2_73_cuda_kernel_int_regs, &threads, &stream, 0);
         }
         else 
         {
            loop_cuda_get_config(loop_ref, 0, loop_test2_73_cuda_kernel_llong_regs, &threads, &stream, 0);
         }
         loop_fill_bounds_(loop_ref, idxL, idxH, 0);
         blocksS[1] = (idxH[1] - idxL[1] + threads.x) / threads.x;
         blocksS[0] = blocksS[1] * ((idxH[0] - idxL[0] + threads.y) / threads.y);
         overallBlocks = blocksS[0];
         restBlocks = overallBlocks;
         addBlocks = 0;
         blocks = dim3(1, 1, 1);
         maxBlocks = loop_cuda_get_device_prop(loop_ref, CUDA_MAX_GRID_X);

// GPU execution
         while (restBlocks > 0)
         {
            if (restBlocks <= maxBlocks) 
            {
               blocks = restBlocks;
            }
            else 
            {
               blocks = maxBlocks;
            }
            if (idxTypeInKernel == rt_INT) 
            {
               loop_test2_73_cuda_kernel_int<<<blocks, threads, 0, stream>>>((float *)b_base, d_b[1], (float *)a_base, d_a[1], idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks);
            }
            else 
            {
               loop_test2_73_cuda_kernel_llong<<<blocks, threads, 0, stream>>>((float *)b_base, d_b[1], (float *)a_base, d_a[1], idxL[0], idxH[0], idxL[1], idxH[1], blocksS[1], addBlocks);
            }
            addBlocks += blocks.x;
            restBlocks -= blocks.x;
         }
      }

}
